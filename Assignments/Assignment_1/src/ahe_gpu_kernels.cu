#include "hip/hip_runtime.h"
// Implement kernels here (Note: delete sample code below)
#include <stdio.h>
#include <iostream>
#include<fstream>
#include "ahe_gpu.h"
#include "utils.h"
#include<map>

template<typename T>
void print_mappings(T* d_mappings, int mapping_size, std::string filename){
    T* mappings = new T[mapping_size];
    int mapping_size_bytes = mapping_size*sizeof(T);
    SAFE_CALL(hipMemcpy, (void*)mappings, (void*)d_mappings,mapping_size_bytes, hipMemcpyDeviceToHost)
    
    std::map<int, int> counts;
    std::cout << "Printing mappings on GPU\n";
    for(int i = 0; i < mapping_size; i++){
        int i__ = mappings[i];
        if(counts.find(i__) == counts.end()){
            counts[i__] = 1;
            continue;
        }
        counts[i__]++;
    }
    std::ofstream MyFile(filename + ".txt");
	for(auto& p : counts){
		MyFile << p.first << ':' << p.second << '\n';
	}
    MyFile << std::endl;
	MyFile.close();
    delete []mappings;   
}

__constant__ int Width[1];
__constant__ int Heigth[1];

__constant__ int Ntiles_x[1];
__constant__ int Ntiles_y[1];
__constant__ int Numtiles[1];

__global__ void ahe_get_PDF(unsigned char* img_in, int* pdf){
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    int width = Width[0];
    if ( x < width && y < Heigth[0]){
        int tile_i = x / TILE_SIZE_X;
		int tile_j = y / TILE_SIZE_Y;
		int offset = 256*(tile_i + tile_j*Ntiles_x[0]);
        atomicAdd(&pdf[offset + img_in[x+y*width]], 1);
    }
}

void get_pdf( unsigned char* d_img_in, int* d_pdf, int width, int height){
    
    int num_threads_x = 32;
    int num_threads_y = 32;
    dim3 block_shape = dim3( num_threads_x, num_threads_y ,1);  

    int num_blocks_x = (width / num_threads_x) + 1; 
    int num_blocks_y = (height / num_threads_y) + 1;

    dim3 grid_shape = dim3( num_blocks_x, num_blocks_y , 1); 

    printf("\nStep 1 (Get pdf): Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n",
    grid_shape.x, grid_shape.y, grid_shape.z, block_shape.x, block_shape.y, block_shape.z);
    
    ahe_get_PDF<<< grid_shape, block_shape>>>(d_img_in, d_pdf);
}

__global__ void ahe_get_mappings(int*pdf_global, unsigned char* mappings){
    __shared__ int cdf[256];
    __shared__ int pdf[256];

    int i = threadIdx.x ;
    int global_i = i + blockIdx.x*blockDim.x;
    if ( global_i < Numtiles[0]*256){
		pdf[i] = pdf_global[global_i];
        __syncthreads();
        //Very naive. Can do better ?
        int cdf_i = pdf[i];
        for(int j = 0; j < i; j++){
            cdf_i += pdf[j];
        }
        cdf[i] = cdf_i;
        __syncthreads();
        int cdf_min = PIXELS_PER_TILE+1; // minimum non-zero value of the CDF
		for(int j=0; j<256; j++){
		    if(cdf[j] != 0) {
				cdf_min = cdf[j]; 
				break;
			}
		}
        float val = (255.0 * float(cdf[i] - cdf_min)/float(PIXELS_PER_TILE - cdf_min));
		mappings[global_i] = (unsigned char)val;
    }
}

void get_mappings( int* d_pdf, unsigned char* d_mappings, int num_tiles){
    
    int num_threads = 256;  
    int num_blocks = num_tiles;

    printf("Step 2 (Get mappings): Grid : {%d} blocks. Blocks : {%d} threads.\n",
    num_blocks, num_threads);
    
    ahe_get_mappings<<< num_blocks, num_threads>>>(d_pdf, d_mappings);
}


extern "C" void run_ahe_GPU(unsigned char* img_in, unsigned char* img_out, int width, int height){
    
	int ntiles_x = width / TILE_SIZE_X;
	int ntiles_y = height / TILE_SIZE_Y;
    int num_tiles = ntiles_x*ntiles_y;

    SAFE_CALL( hipMemcpyToSymbol, Width, &width, sizeof(width))
    SAFE_CALL( hipMemcpyToSymbol, Heigth, &height, sizeof(height))
    SAFE_CALL( hipMemcpyToSymbol, Ntiles_x, &ntiles_x, sizeof(ntiles_x))
    SAFE_CALL( hipMemcpyToSymbol, Ntiles_y, &ntiles_y, sizeof(ntiles_y))
    SAFE_CALL( hipMemcpyToSymbol, Numtiles, &num_tiles, sizeof(num_tiles))

    int img_size = height*width;
    

    int img_size_bytes = img_size*sizeof(unsigned char);
    unsigned char *d_img_in;
    SAFE_CALL( hipMalloc, (void**)&d_img_in, img_size_bytes)
    SAFE_CALL( hipMemcpy, (void*)d_img_in, (void*)img_in, img_size_bytes, hipMemcpyHostToDevice)
    
    int pdf_size = num_tiles*256;
    int pdf_size_bytes = pdf_size*sizeof(int);
    int* d_pdf;
    SAFE_CALL( hipMalloc, (void**)&d_pdf, pdf_size_bytes)
    SAFE_CALL( hipMemset, (void*)d_pdf, 0, pdf_size_bytes )
    get_pdf(d_img_in, d_pdf, width, height);
    SAFE_CALL( hipDeviceSynchronize)
    
    unsigned char* d_mappings;
    int mappings_size = num_tiles*256;
    int mappings_size_bytes = pdf_size*sizeof(unsigned char);
    SAFE_CALL( hipMalloc, (void**)&d_mappings, mappings_size_bytes)
    get_mappings(d_pdf, d_mappings, num_tiles);
    SAFE_CALL( hipDeviceSynchronize )

    print_mappings(d_mappings, mappings_size, "mappings_GPU");
    SAFE_CALL( hipFree, d_img_in)
    SAFE_CALL( hipFree, d_mappings)
 
}